#include "stdio.h"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>


#define THREADS 1024

#define gpu_error_check(ans) { gpu_assert((ans), __FILE__, __LINE__); }
inline void gpu_assert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void swap(int* array, int first, int second)
{   
    int tmp = array[first];
    array[first] = array[second];
    array[second] = tmp;
}

__device__
void swap_gpu(int* array, int first, int second)
{   
    int tmp = array[first];
    array[first] = array[second];
    array[second] = tmp;
}

__global__
void bitonic_exchange_gpu(int* dev_values, int depth, unsigned long step)
{
    /* Sorting partners: i and pair_for_i */
    unsigned int i, pair_for_i; 
    
    /* Orient tells in which part of bitonic (sub-)sequence elements are (descending or ascending) */
    unsigned int orient_i, orient_pair_for_i; 

    i = threadIdx.x + blockDim.x * blockIdx.x;
    pair_for_i = i + depth;

    orient_pair_for_i = pair_for_i & step;
    orient_i = i & step;

    /* 
        If current array[i] is the second for other array[j] (i<j) so we just do nothing
        It can be seen if for some a[i], a[pair_for_i] located in other bitonic (sub-)sequence
        For example, a[i] in ascending part, a[pair_for_i] in descending
    */
    if (orient_i != 0 && orient_pair_for_i == 0 
        || orient_i == 0 && orient_pair_for_i != 0)
    {
        return;
    }

    if (orient_i == 0) 
    {
        /* Sort ascending */
        if (dev_values[i]>dev_values[pair_for_i])
        {
            swap_gpu(dev_values, i, pair_for_i);
        }
	}
    else 
    {
		/* Sort descending */
        if (dev_values[i]<dev_values[pair_for_i])
        {
        	swap_gpu(dev_values, i, pair_for_i);
        }
	}
}

void bitonic_sort_gpu(int* array, unsigned long size)
{
    size_t size_mem_array = size * sizeof(int);
    int* array_gpu;

    gpu_error_check(hipMalloc(&array_gpu, size_mem_array));
    gpu_error_check(hipMemcpy(array_gpu, array, size_mem_array, hipMemcpyHostToDevice));

    dim3 blocks = (size < THREADS) ? size : size / THREADS;
    dim3 threadsPerBlock = (size < THREADS) ? 1 : THREADS;

    for (int step = 2; step <= size; step <<= 1)
    {
        for (int depth = step >> 1; depth >= 1; depth >>= 1)
        {
            bitonic_exchange_gpu<<<blocks, threadsPerBlock>>>(array_gpu, depth , step);
        }
	}
	
	gpu_error_check(hipMemcpy(array, array_gpu, size_mem_array, hipMemcpyDeviceToHost));
	hipFree(&array_gpu);
}


void bitonic_exchange(int* array, int depth, int step, unsigned long size)
{
    for (int i = 0; i < size; i++)
    {
        unsigned int pair_for_i;
        unsigned int orient_i, orient_pair_for_i;

        pair_for_i = i + depth;

        orient_pair_for_i = pair_for_i & step;
        orient_i = i & step;

        if (orient_i != 0 && orient_pair_for_i == 0 
            || orient_i == 0 && orient_pair_for_i != 0)
        {
            continue;
        }
        
        if (orient_i == 0)
        {
            if (array[i] > array[pair_for_i])
            {
                swap(array, i, pair_for_i);
            }
        } 
        else 
        {
            if (array[i] < array[pair_for_i])
            {
                swap(array, i, pair_for_i);
            }
        }
    }
}

void bitonic_sort(int* array, unsigned long size)
{   
    for (int step = 2; step <= size; step <<= 1)
    {
        for (int j = step >> 1 ; j >= 1; j >>= 1)
        {
            bitonic_exchange(array, j, step, size);
        }
    }
}