
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"

void print_array(int* array, int size)
{
    for (int i = 0; i < size; ++i)
    {
        printf("%d ", array[i]);
    }
}

void generate_random_array(int* arr, int size)
{
    for (int i = 0; i < size; ++i)
    {
        arr[i] = rand() % (size * 10);
    }
}