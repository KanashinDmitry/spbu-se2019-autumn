#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#include "string.h"
#include "bitonic_sorts.cuh"
#include "utils.cuh"

#define AMOUNT_MEASUREMENTS 20

int main(char* args)
{
    clock_t start;
    unsigned long upperBound = 1024 << 13;
    
    for (unsigned size = 1024; size <= upperBound; size <<= 1)
    {        
        srand(time(NULL));
        
        size_t size_mem_array = size*sizeof(int);

        int* array = (int*) malloc(size_mem_array);
        int* temp_array = (int*) malloc(size_mem_array);
        
        double timeGPU = 0;
        double timeCPU = 0;

        generate_random_array(array, size);
        
        for (int i = 0; i < 20; i++)
        {
            memcpy(temp_array, array, size_mem_array);
            
            start = clock();
            bitonic_sort(temp_array, size);
            timeCPU += ((double) (clock() - start));

            memcpy(temp_array, array, size_mem_array);

            start = clock();
            bitonic_sort_gpu(temp_array, size);
            timeGPU += ((double) (clock() - start));
        }

        double avg_timeCPU = timeCPU  / CLOCKS_PER_SEC / AMOUNT_MEASUREMENTS;
        double avg_timeGPU = timeGPU  / CLOCKS_PER_SEC / AMOUNT_MEASUREMENTS;

        printf("%ld %f %f\n", size, avg_timeCPU, avg_timeGPU);
        
        free(array);
        free(temp_array);
    }

    return 0;
}